#ifndef _DELITE_CUDA_
#define _DELITE_CUDA_

#include "DeliteCuda.h"

using namespace std;

list<void*>* lastAlloc = new list<void*>();
queue<FreeItem>* freeList = new queue<FreeItem>();
map<void*,list<void*>*>* cudaMemoryMap = new map<void*,list<void*>*>();

void freeCudaMemory(FreeItem item) {
    list<void*>::iterator iter;
    for (iter = item.keys->begin(); iter != item.keys->end(); iter++) {
        //cout << "object ref: " << (long) *iter << endl;
        if(cudaMemoryMap->find(*iter) != cudaMemoryMap->end()) {
        	list<void*>* freePtrList = cudaMemoryMap->find(*iter)->second;
       		list<void*>::iterator iter2;
        	for (iter2 = freePtrList->begin(); iter2 != freePtrList->end(); iter2++) {
            	void* freePtr = *iter2;
            	hipFree(freePtr);
            	//if (hipFree(freePtr) != hipSuccess)
            	//    cout << "bad free pointer: " << (long) freePtr << endl;
            	//else
                	//cout << "freed successfully: " << (long) freePtr << endl;
       		}
        	cudaMemoryMap->erase(*iter);
        	delete freePtrList;
        	free(*iter);
		}
    }
    delete item.keys;
}

void DeliteCudaMalloc(void** ptr, size_t size) {

    while (freeList->size() != 0) {
	FreeItem item = freeList->front();
    	if (hipEventQuery(item.event) != hipSuccess) {
	    break;
	}
	freeList->pop();
	hipEventDestroy(item.event);
	freeCudaMemory(item);
    }

    while (hipMalloc(ptr, size) != hipSuccess) {
        if (freeList->size() == 0) {
	    cout << "FATAL: Insufficient device memory" << endl;
	    exit(-1);
	}
	FreeItem item = freeList->front();
        freeList->pop();

        while (hipEventQuery(item.event) != hipSuccess) {
            hipEventSynchronize(item.event);
        }
        hipEventDestroy(item.event);
	freeCudaMemory(item);
    }

    lastAlloc->push_back(*ptr);
}

/*
void DeliteCudaMalloc(void** ptr, size_t size) {
    while (freeList->size() > 0) {
	    FreeItem item = freeList->front();
 	        freeList->pop();
		
		while (hipEventQuery(item.event) != hipSuccess) {
			hipEventSynchronize(item.event);
		}
		hipEventDestroy(item.event);
		
		list<void*>::iterator iter;
		for (iter = item.keys->begin(); iter != item.keys->end(); iter++) {
			//cout << "object ref: " << (long) *iter << endl;
			list<void*>* freePtrList = cudaMemoryMap->find(*iter)->second;	
			list<void*>::iterator iter2;
			for (iter2 = freePtrList->begin(); iter2 != freePtrList->end(); iter2++) {
				void* freePtr = *iter2;
				if (hipFree(freePtr) != hipSuccess)
					cout << "bad free pointer: " << (long) freePtr << endl;
				else
					cout << "freed successfully: " << (long) freePtr << endl;
			}
			cudaMemoryMap->erase(*iter);
			delete freePtrList;
			free(*iter);
		}
		delete item.keys;
	}

	if (hipMalloc(ptr, size) != hipSuccess) {
		cout << "FATAL: cuda malloc failed unexpectedly" << endl;
		exit(-1);
	}
	else
		cout << "allocated successfully: " << (long) *ptr << endl;
	
	lastAlloc->push_back(*ptr);
} */

char* bufferStart = 0;
size_t bufferSize = 5368709120/4;
char* bufferEnd;
char* bufferCurrent;

void hostInit() {
	hipHostAlloc(&bufferStart, bufferSize, hipHostMallocDefault);
	bufferEnd = bufferStart + bufferSize;
	bufferCurrent = bufferStart;
}

void DeliteCudaMallocHost(void** ptr, size_t size) {
	if (bufferStart == 0) hostInit();
	if ((bufferCurrent + size) > bufferEnd)
		bufferCurrent = bufferStart;
	*ptr = bufferCurrent;
	bufferCurrent += size;
}

void DeliteCudaMemcpyHtoDAsync(void* dptr, void* sptr, size_t size) {
	hipMemcpyAsync(dptr, sptr, size, hipMemcpyHostToDevice, h2dStream);
}

void DeliteCudaMemcpyDtoHAsync(void* dptr, void* sptr, size_t size) {
	hipMemcpyAsync(dptr, sptr, size, hipMemcpyDeviceToHost, d2hStream);
	hipStreamSynchronize(d2hStream);
}

void DeliteCudaMemcpyDtoDAsync(void *dptr, void* sptr, size_t size) {
	hipMemcpyAsync(dptr, sptr, size, hipMemcpyDeviceToDevice, kernelStream);
}

void DeliteCudaMemset(void *ptr, int value, size_t count) {
	hipMemset(ptr,value,count);
}

#endif
